#include "hip/hip_runtime.h"
/**
 * BUDE CUDA kernel file
 **/

#include <float.h>
#include <stdio.h>
#include "shared.h"

// Numeric constants
#define ZERO    0.0f
#define QUARTER 0.25f
#define HALF    0.5f
#define ONE     1.0f
#define TWO     2.0f
#define FOUR    4.0f
#define CNSTNT 45.0f

#define HBTYPE_F 70
#define HBTYPE_E 69

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

// The data structure for one atom - 16 bytes

typedef struct
{
  float x, y, z, w;
} Transform;

#define HARDNESS 38.0f
#define NPNPDIST  5.5f
#define NPPDIST   1.0f

__global__ void fasten_main(const int natlig,
    const int natpro,
    const Atom* protein_molecule,
    const Atom* ligand_molecule,
    const float* transforms_0,
    const float* transforms_1,
    const float* transforms_2,
    const float* transforms_3,
    const float* transforms_4,
    const float* transforms_5,
    float* etotals,
    const FFParams* forcefield,
    const int num_atom_types,
    const int numTransforms);

  extern "C"
void runCUDA(float* results)
{
  printf("\nRunning CUDA\n");

  hipSetDevice(0);
  CudaSafeCall(hipMalloc((void**)&_cuda.d_protein, params.natpro*sizeof(Atom)));
  hipDeviceSynchronize();
  CudaSafeCall(hipMemcpy(_cuda.d_protein, params.protein, params.natpro*sizeof(Atom), hipMemcpyHostToDevice));
  hipDeviceSynchronize();

  CudaSafeCall(hipMalloc((void**)&_cuda.d_ligand, params.natlig*sizeof(Atom)));
  hipDeviceSynchronize();
  CudaSafeCall(hipMemcpy(_cuda.d_ligand, params.ligand, params.natlig*sizeof(Atom), hipMemcpyHostToDevice));
  hipDeviceSynchronize();

  CudaSafeCall(hipMalloc((void**)&_cuda.d_forcefield, params.ntypes*sizeof(FFParams)));
  hipDeviceSynchronize();
  CudaSafeCall(hipMemcpy(_cuda.d_forcefield, params.forcefield, params.ntypes*sizeof(FFParams), hipMemcpyHostToDevice));
  hipDeviceSynchronize();

  CudaSafeCall(hipMalloc((void**)&_cuda.d_results, params.nposes*sizeof(float)));
  hipDeviceSynchronize();

  for(int ii = 0; ii < 6; ++ii)
  {
    CudaSafeCall(hipMalloc((void**)&_cuda.d_poses[ii], params.nposes*sizeof(float)));
    hipDeviceSynchronize();
    CudaSafeCall(hipMemcpy(_cuda.d_poses[ii], params.poses[ii], params.nposes*sizeof(float), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
  }

  size_t global = ceil(params.nposes/(double)_cuda.posesPerWI);
  global = ceil(global/(double)_cuda.wgsize);
  size_t local  = _cuda.wgsize;
  size_t shared = params.ntypes * sizeof(FFParams);

  hipDeviceSynchronize();

  double start = getTimestamp();

  for(int ii = 0; ii < params.iterations; ++ii)
  {
    fasten_main<<<global, local, shared>>>(
        params.natlig, 
        params.natpro,
        _cuda.d_protein,
        _cuda.d_ligand,
        _cuda.d_poses[0],
        _cuda.d_poses[1],
        _cuda.d_poses[2],
        _cuda.d_poses[3],
        _cuda.d_poses[4],
        _cuda.d_poses[5],
        _cuda.d_results,
        _cuda.d_forcefield,
        params.ntypes,
        params.nposes);
    CudaSafeCall(hipPeekAtLastError());
  }

  hipDeviceSynchronize();

  double end = getTimestamp();

  CudaSafeCall(hipMemcpy(results, _cuda.d_results, params.nposes*sizeof(float), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  printTimings(start, end, _cuda.posesPerWI);
}

__device__ void compute_transformation_matrix(const float transform_0,
    const float transform_1,
    const float transform_2,
    const float transform_3,
    const float transform_4,
    const float transform_5,
    Transform* transform)
{
  const float sx = sin(transform_0);
  const float cx = cos(transform_0);
  const float sy = sin(transform_1);
  const float cy = cos(transform_1);
  const float sz = sin(transform_2);
  const float cz = cos(transform_2);

  transform[0].x = cy*cz;
  transform[0].y = sx*sy*cz - cx*sz;
  transform[0].z = cx*sy*cz + sx*sz;
  transform[0].w = transform_3;
  transform[1].x = cy*sz;
  transform[1].y = sx*sy*sz + cx*cz;
  transform[1].z = cx*sy*sz - sx*cz;
  transform[1].w = transform_4;
  transform[2].x = -sy;
  transform[2].y = sx*cy;
  transform[2].z = cx*cy;
  transform[2].w = transform_5;
}

__global__ void fasten_main(const int natlig,
    const int natpro,
    const Atom* __restrict protein_molecule,
    const Atom* __restrict ligand_molecule,
    const float* __restrict transforms_0,
    const float* __restrict transforms_1,
    const float* __restrict transforms_2,
    const float* __restrict transforms_3,
    const float* __restrict transforms_4,
    const float* __restrict transforms_5,
    float* __restrict etotals,
    const FFParams* global_forcefield,
    const int num_atom_types,
    const int numTransforms)
{
  // Get index of first TD
  int ix = blockIdx.x*blockDim.x*NUM_TD_PER_THREAD + threadIdx.x;

  // Have extra threads do the last member intead of return.
  // A return would disable use of barriers, so not using return is better
  ix = ix < numTransforms ? ix : numTransforms - NUM_TD_PER_THREAD;

#ifdef USE_SHARED
  extern __shared__ FFParams forcefield[];
  if(ix < num_atom_types)
  {
    forcefield[ix] = global_forcefield[ix];
  }
#else
  const FFParams* forcefield = global_forcefield;
#endif

  // Compute transformation matrix to private memory
  float etot[NUM_TD_PER_THREAD];
  Transform transform[NUM_TD_PER_THREAD][3];
  const int lsz = blockDim.x;
  for (int i = 0; i < NUM_TD_PER_THREAD; i++)
  {
    int index = ix + i*lsz;
    compute_transformation_matrix(
        transforms_0[index],
        transforms_1[index],
        transforms_2[index],
        transforms_3[index],
        transforms_4[index],
        transforms_5[index],
        transform[i]);
    etot[i] = ZERO;
  }

#ifdef USE_SHARED
  __syncthreads();
#endif

  // Loop over ligand atoms
  int il = 0;
  do
  {
    // Load ligand atom data
    const Atom l_atom = ligand_molecule[il];

    const FFParams l_params = forcefield[l_atom.index];
    const bool lhphb_ltz = l_params.hphb<ZERO;
    const bool lhphb_gtz = l_params.hphb>ZERO;

    float3 lpos[NUM_TD_PER_THREAD];
    const float4 linitpos = make_float4(l_atom.x,l_atom.y,l_atom.z,ONE);
    for (int i = 0; i < NUM_TD_PER_THREAD; i++)
    {
      // Transform ligand atom
      lpos[i].x = transform[i][0].w + linitpos.x*transform[i][0].x + 
        linitpos.y*transform[i][0].y + linitpos.z*transform[i][0].z;
      lpos[i].y = transform[i][1].w + linitpos.x*transform[i][1].x + 
        linitpos.y*transform[i][1].y + linitpos.z*transform[i][1].z;
      lpos[i].z = transform[i][2].w + linitpos.x*transform[i][2].x + 
        linitpos.y*transform[i][2].y + linitpos.z*transform[i][2].z;
    }

    // Loop over protein atoms
    int ip = 0;
    do
    {
      // Load protein atom data
      const Atom p_atom = protein_molecule[ip];

      const FFParams p_params = forcefield[p_atom.index];

      const float radij   = p_params.radius + l_params.radius;
      const float r_radij = 1.0f/radij;

      const float elcdst  = (p_params.hbtype==HBTYPE_F && l_params.hbtype==HBTYPE_F) ? FOUR    : TWO;
      const float elcdst1 = (p_params.hbtype==HBTYPE_F && l_params.hbtype==HBTYPE_F) ? QUARTER : HALF;
      const bool type_E   = ((p_params.hbtype==HBTYPE_E || l_params.hbtype==HBTYPE_E));

      const bool phphb_ltz = p_params.hphb<ZERO;
      const bool phphb_gtz = p_params.hphb>ZERO;
      const bool phphb_nz  = p_params.hphb!=ZERO;
      const float p_hphb   = p_params.hphb * (phphb_ltz && lhphb_gtz ? -ONE : ONE);
      const float l_hphb   = l_params.hphb * (phphb_gtz && lhphb_ltz ? -ONE : ONE);
      const float distdslv = (phphb_ltz ? (lhphb_ltz ? NPNPDIST : NPPDIST) : (lhphb_ltz ? NPPDIST : -FLT_MAX));
      const float r_distdslv = 1.0f/distdslv;

      const float chrg_init = l_params.elsc * p_params.elsc;
      const float dslv_init = p_hphb + l_hphb;

      for (int i = 0; i < NUM_TD_PER_THREAD; i++)
      {
        // Calculate distance between atoms
        const float x      = lpos[i].x - p_atom.x;
        const float y      = lpos[i].y - p_atom.y;
        const float z      = lpos[i].z - p_atom.z;
        const float distij = sqrt(x*x + y*y + z*z);

        // Calculate the sum of the sphere radii
        const float distbb = distij - radij;
        const bool  zone1  = (distbb < ZERO);

        // Calculate steric energy
        etot[i] += (ONE - (distij*r_radij)) * (zone1 ? 2*HARDNESS : ZERO);

        // Calculate formal and dipole charge interactions
        float chrg_e = chrg_init * ((zone1 ? 1 : (ONE - distbb*elcdst1)) 
            * (distbb<elcdst ? 1 : ZERO));
        const float neg_chrg_e = -fabs(chrg_e);
        chrg_e = type_E ? neg_chrg_e : chrg_e;
        etot[i] += chrg_e*CNSTNT;

        // Calculate the two cases for Nonpolar-Polar repulsive interactions
        const float coeff  = (ONE - (distbb*r_distdslv));
        float dslv_e = dslv_init * ((distbb<distdslv && phphb_nz) ? 1 : ZERO);
        dslv_e *= (zone1 ? 1 : coeff);
        etot[i] += dslv_e;
      }
    } 
    while (++ip < natpro); // loop over protein atoms
  } 
  while (++il < natlig); // loop over ligand atoms

  // Write results
  const int td_base = blockIdx.x*blockDim.x*NUM_TD_PER_THREAD + threadIdx.x;
  if (td_base < numTransforms)
  {
    for (int i = 0; i < NUM_TD_PER_THREAD; i++)
    {
      etotals[td_base+i*blockDim.x] = etot[i]*HALF;
    }
  }
} //end of fasten_main



